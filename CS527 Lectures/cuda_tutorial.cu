#include "hip/hip_runtime.h"

#include <iostream>
#include <conio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <ctime>
#include <stdlib.h>
#include <chrono>

#define BLOCK_SIZE 128

using namespace std;
using namespace std::chrono;

__global__ void PrefixSum(int* a, int n)
{
	for (int i = 1; i < n; i++) {
		a[i] += a[i - 1];
	}
}

int main()
{
	srand(time(NULL));
	for (int n = BLOCK_SIZE; n <= 128 * BLOCK_SIZE; n += BLOCK_SIZE) {
		//cout << "n is " << n << '\n';
		int* h_a = new int[n];
		for (int i = 0; i < n; i++)
			h_a[i] = 1;

		int* d_a;
		if (hipMalloc(&d_a, sizeof(int) * n) != hipSuccess) {
			cout << "Failed to allocate global memory in GPU.\n";
			delete[] h_a;
			return 0;
		}

		if (hipMemcpy(d_a, h_a, sizeof(int) * n, hipMemcpyHostToDevice) != hipSuccess) {
			cout << "Failed to copy to global memory in GPU.\n";
			hipFree(d_a);
			delete[] h_a;
			return 0;
		}

		auto start = high_resolution_clock::now();
		PrefixSum <<<1, 1>>> (d_a, n);
		hipDeviceSynchronize();
		auto stop = high_resolution_clock::now();

		if (hipMemcpy(h_a, d_a, sizeof(int) * n, hipMemcpyDeviceToHost) != hipSuccess) {
			hipFree(d_a);
			delete[] h_a;
			cout << "Failed to copy to main memory in CPU.\n";
			return 0;
		}

		for (int i = 0; i < n; i++) {
			if (h_a[i] != i + 1) {
				cout << "FAIL";
				break;
			}
		}
		//cout << '\n';

		auto duration = duration_cast<microseconds>(stop - start);
		cout << "n = " << n << ", t = " << duration.count() << '\n';
	}
	return 0;

}




/*
#include <iostream>
#include <conio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <ctime>
#include <stdlib.h>
#include <chrono>

#define BLOCK_SIZE 128

using namespace std;
using namespace std::chrono;



__device__ void UpSweep(int* shared_memory, int i)
{
	// Traversing the prefix tree from bottom to top
	for (int d = 1; d < BLOCK_SIZE; d *= 2) {
		// Performing additions at depth d in parallel and synchronizing
		if (i % (2 * d) == 0)
			shared_memory[i + 2 * d - 1] = shared_memory[i + d - 1] + shared_memory[i + 2 * d - 1];
		__syncthreads();
	}
}

__device__ void DownSweep(int* shared_memory, int i)
{
	// Traversing the prefix tree from top to bottom
	for (int d = BLOCK_SIZE / 4; d >= 1; d /= 2) {
		// Performing additions at depth d in parallel and synchronizing
		if(i % (2 * d) == 0 && (i + 2 * d - 1 + d) < BLOCK_SIZE)
			shared_memory[i + 2 * d - 1 + d] += shared_memory[i + 2 * d - 1];
		__syncthreads();
	}
}


__global__ void Aggregate(int* a, int n)
{
	// Initializing shared memory half for array elements and half for last elements of each block
	__shared__ int shared_memory_2[2 * BLOCK_SIZE];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	shared_memory_2[threadIdx.x] = 0;
	shared_memory_2[BLOCK_SIZE + threadIdx.x] = 0;
	__syncthreads();

	// Copying array elements to shared memory
	if (i < n)
		shared_memory_2[threadIdx.x] = a[i];
	__syncthreads();

	// Copying last element of each block to shared memory
	int j = (blockIdx.x + 1) * BLOCK_SIZE - 1;
	if(j < n)
		shared_memory_2[threadIdx.x + BLOCK_SIZE] = a[j];
	__syncthreads();

	// Performing upsweep and downsweep steps
	UpSweep(shared_memory_2 + BLOCK_SIZE, threadIdx.x);
	DownSweep(shared_memory_2 + BLOCK_SIZE, threadIdx.x);

	// Adding the resulting prefix sum to the respective array elements in shared memory
	if (blockIdx.x >= 1) {
		shared_memory_2[threadIdx.x] += shared_memory_2[BLOCK_SIZE + blockIdx.x - 1];
	}
	__syncthreads();
	
	// Copying the elements of shared memory back to array in global memory
	a[i] = shared_memory_2[threadIdx.x];
	__syncthreads();
}


__global__ void ParallelPrefixSum(int* a, int n)
{
	// Initializing shared memory of one block size section of the array
	__shared__ int shared_memory[BLOCK_SIZE];
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	shared_memory[threadIdx.x] = 0;
	__syncthreads();
	
	if (i < n) {
		// Copying the elements of array into shared memory
		shared_memory[threadIdx.x] = a[i];
		__syncthreads();

		// Performing the upsweep and downsweep steps
		UpSweep(shared_memory, threadIdx.x);
		DownSweep(shared_memory, threadIdx.x);
		
		// Copying the elements of shared memory back to array in global memory
		a[i] = shared_memory[threadIdx.x];
	}	
}

int main()
{
	srand(time(NULL));
	
	for (int n = BLOCK_SIZE; n <= 128 * BLOCK_SIZE; n += BLOCK_SIZE) {
		// Declaration and Allocation of memory in host
		int* h_a = new int[n];
		for (int i = 0; i < n; i++)
			h_a[i] = 1;

		// Declaration and Allocation of memory in device
		int* d_a;
		if (hipMalloc(&d_a, sizeof(int) * n) != hipSuccess) {
			cout << "Failed to allocate global memory in GPU.\n";
			delete[] h_a;
			return 0;
		}

		// Data transfer from host to device memory
		if (hipMemcpy(d_a, h_a, sizeof(int) * n, hipMemcpyHostToDevice) != hipSuccess) {
			cout << "Failed to copy to global memory in GPU.\n";
			hipFree(d_a);
			delete[] h_a;
			return 0;
		}

		// Kernels execute
		auto start = high_resolution_clock::now();
		ParallelPrefixSum <<<n / BLOCK_SIZE + 1, BLOCK_SIZE >>> (d_a, n);
		hipDeviceSynchronize();
		Aggregate <<<n / BLOCK_SIZE + 1, BLOCK_SIZE >>> (d_a, n);
		hipDeviceSynchronize();
		auto stop = high_resolution_clock::now();

		// Data transfers back to host
		if (hipMemcpy(h_a, d_a, sizeof(int) * n, hipMemcpyDeviceToHost) != hipSuccess) {
			hipFree(d_a);
			delete[] h_a;
			cout << "Failed to copy to main memory in CPU.\n";
			return 0;
		}

		// Checking for  correctness
		for (int i = 0; i < n; i++) {
			//cout << h_a[i] << ' ';
			if (h_a[i] != i + 1) {
				cout << "FAIL";
				break;
			}
		}
		cout << '\n';

		// Printing the output
		auto duration = duration_cast<microseconds>(stop - start);
		cout << "n = " << n << ", t = " << duration.count() << '\n';
	}
	return 0;

}
*/